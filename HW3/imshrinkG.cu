#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <iostream>
#include <ctype.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define	CEIL(a,b)		((a+b-1)/b)
#define SWAP(a,b,t)		t=b; b=a; a=t;
#define DATAMB(bytes)			(bytes/1024/1024)
#define DATABW(bytes,timems)	((float)bytes/(timems * 1.024*1024.0*1024.0))

typedef unsigned char uch;
typedef unsigned long ul;
typedef unsigned int  ui;

uch *image, *imgCpy;
uch *imageG, *imgCpyG, *outG;

struct ImgProp{
	int Hpixels;
	int Vpixels;
	uch HeaderInfo[54];
	ul Hbytes;
} ip;

#define	IPHB		ip.Hbytes
#define	IPH			ip.Hpixels
#define	IPV			ip.Vpixels
#define	IMAGESIZE	(IPHB*IPV)
#define	IMAGEPIX	(IPH*IPV)

__global__
void *shrinkG(uch* dest, uch* source, ui skipC, ui skipR) {
	ui ThPerBlk = blockDim.x;
	ui myBid = blockIdx.x;
	ui myTid = threadIdx.x;
	ui myRow = blockIdx.y;
	ui myIndex = (myBid*ThPerBlk + myTid) * skipR * IPHB;
	if (myIndex > IMAGESIZE)
		return;
	ui destIndex = (myBid*ThPerBlk + myTid) * IPHB;
	while (myIndex < (myIndex+IPHB-2)) {
		dest[destIndex] = source[myIndex];
		dest[destIndex+1] = source[myIndex+1];
		dest[destIndeex+2] = source[myIndex+2];
		destIndex += 3;
		myIndex += (skipC*3);
	}
}

uch *ReadBMPlin(char* fn)
{
	static uch *Img;
	FILE* f = fopen(fn, "rb");
	if (f == NULL){	printf("%s NOT FOUND\n", fn);	exit(EXIT_FAILURE); }

	uch HeaderInfo[54];
	fread(HeaderInfo, sizeof(uch), 54, f); // read the 54-byte header
	// extract image height and width from header
	int width = *(int*)&HeaderInfo[18];			ip.Hpixels = width;
	int height = *(int*)&HeaderInfo[22];		ip.Vpixels = height;
	int RowBytes = (width * 3 + 3) & (~3);		ip.Hbytes = RowBytes;
	//save header for re-use
	memcpy(ip.HeaderInfo, HeaderInfo,54);
	printf("Input File name: %17s  (%u x %u)   File Size=%u\n", fn, 
			ip.Hpixels, ip.Vpixels, IMAGESIZE);
	// allocate memory to store the main image (1 Dimensional array)
	Img  = (uch *)malloc(IMAGESIZE);
	if (Img == NULL) return Img;      // Cannot allocate memory
	// read the image from disk
	fread(Img, sizeof(uch), IMAGESIZE, f);
	fclose(f);
	return Img;
}


// Write the 1D linear-memory stored image into file.
void WriteBMPlin(uch *Img, char* fn)
{
	FILE* f = fopen(fn, "wb");
	if (f == NULL){ printf("FILE CREATION ERROR: %s\n", fn); exit(1); }
	//write header
	fwrite(ip.HeaderInfo, sizeof(uch), 54, f);
	//write data
	fwrite(Img, sizeof(uch), IMAGESIZE, f);
	printf("Output File name: %17s  (%u x %u)   File Size=%u\n", fn, ip.Hpixels, ip.Vpixels, IMAGESIZE);
	fclose(f);
}


int main(int argc, char **argv)
{
	float			totalTime, tfrCPUtoGPU, tfrGPUtoCPU, kernelExecutionTime; // GPU code run times
	hipError_t		cudaStatus, cudaStatus2;
	hipEvent_t		time1, time2, time3, time4;
	ui				BlkPerRow, BlkPerRowInt, BlkPerRowInt2;
	ui				ThrPerBlk = 256, NumBlocks, NB2, NB4, NB8;
	hipDeviceProp_t	GPUprop;
	ul				SupportedKBlocks, SupportedMBlocks, MaxThrPerBlk;
	ui				*imgCpyI, *imageI;
	char			SupportedBlocks[100];
	int				KernelNum=1, skipR, skipC;
	char			KernelName[255];

	if (argc != 5)
		printf("Incorrect call. Correct call:\n");
		printf("./filename in.bmp out.bmp xShrink yShrink\n");
		exit(EXIT_FAILURE);
	}
	
	image = ReadBMPlin(argv[1]);
	if (image == NULL){
		printf("Cannot allocate memory for the input image...\n");
		exit(EXIT_FAILURE);
	}
	skipC = atoi(argv[3]);
	skipR = atoi(argv[4]);
	
	NumBlocks = floor(ip.Vpixels/(NumThreads*skipR));
	imgCpy = (uch *)malloc(IMAGESIZE);
	if (imgCpy == NULL){
		free(image);
		printf("Cannot allocate memory for the input image...\n");
		exit(EXIT_FAILURE);
	}

	int NumGPUs = 0;
	hipGetDeviceCount(&NumGPUs);
	if (NumGPUs == 0){
		printf("No CUDA Device is available\n");
		exit(EXIT_FAILURE);
	}
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		exit(EXIT_FAILURE);
	}
	hipGetDeviceProperties(&GPUprop, 0);
	SupportedKBlocks = (ui)GPUprop.maxGridSize[0] * (ui)GPUprop.maxGridSize[1] * (ui)GPUprop.maxGridSize[2] / 1024;
	SupportedMBlocks = SupportedKBlocks / 1024;
	sprintf(SupportedBlocks, "%u %c", (SupportedMBlocks >= 5) ? SupportedMBlocks : SupportedKBlocks, (SupportedMBlocks >= 5) ? 'M' : 'K');
	MaxThrPerBlk = (ui)GPUprop.maxThreadsPerBlock;

	hipEventCreate(&time1);
	hipEventCreate(&time2);
	hipEventCreate(&time3);
	hipEventCreate(&time4);

	hipEventRecord(time1, 0);		// Time stamp at the start of the GPU transfer
	// Allocate GPU buffer for the input and output images
	cudaStatus = hipMalloc((void**)&imageG, IMAGESIZE);
	cudaStatus2 = hipMalloc((void**)&imgCpyG, IMAGESIZE/(skipC*skipR));
	if ((cudaStatus != hipSuccess) || (cudaStatus2 != hipSuccess)) {
		fprintf(stderr, "hipMalloc failed! Can't allocate GPU memory\n");
		exit(EXIT_FAILURE);
	}
	// These are the same pointers as GPUCopyImg and GPUImg, however, casted to an integer pointer
	imgCpyI = (ui *)imgCpyG;
	imageI = (ui *)imageG;

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(imageG, image, IMAGESIZE, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy  CPU to GPU  failed!\n");
		exit(EXIT_FAILURE);
	}

	hipEventRecord(time2, 0);
	
	shrinkG <<<NumBlocks, NumThreads>>> (imgCpyG, imageG, skipC, skipR);
	
	outG = imgCpyG;
	//GPUDataTransfer = 2 * IMAGESIZE;

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching the kernel!\n", cudaStatus);
		exit(EXIT_FAILURE);
	}
	hipEventRecord(time3, 0);

	// Copy output (results) from GPU buffer to host (CPU) memory.
	cudaStatus = hipMemcpy(imgCpy, outG, IMAGESIZE, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy GPU to CPU  failed!\n");
		exit(EXIT_FAILURE);
	}
	hipEventRecord(time4, 0);

	hipEventSynchronize(time1);
	hipEventSynchronize(time2);
	hipEventSynchronize(time3);
	hipEventSynchronize(time4);

	hipEventElapsedTime(&totalTime, time1, time4);
	hipEventElapsedTime(&tfrCPUtoGPU, time1, time2);
	hipEventElapsedTime(&kernelExecutionTime, time2, time3);
	hipEventElapsedTime(&tfrGPUtoCPU, time3, time4);

	cudaStatus = hipDeviceSynchronize();
	//checkError(hipGetLastError());	// screen for errors in kernel launches
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Program failed after hipDeviceSynchronize()!\n");
		free(image);
		free(imgCpy);
		exit(EXIT_FAILURE);
	}
	WriteBMPlin(imgCpy, OutputFileName);
	printf("--------------------------------------------------------------------------\n");
	printf("%s    ComputeCapab=%d.%d  [max %s blocks; %d thr/blk] \n",
		GPUprop.name, GPUprop.major, GPUprop.minor, SupportedBlocks, MaxThrPerBlk);
	printf("--------------------------------------------------------------------------\n");
	printf("%s %s %s %c %u %u  [%u BLOCKS, %u BLOCKS/ROW]\n", ProgName, InputFileName, OutputFileName, Flip, ThrPerBlk, KernelNum, NumBlocks, BlkPerRow);
	printf("--------------------------------------------------------------------------\n");
	printf("%s\n",KernelName);
	printf("--------------------------------------------------------------------------\n");
	printf("CPU->GPU Transfer   =%7.2f ms  ...  %4d MB  ...  %6.2f GB/s\n", tfrCPUtoGPU, DATAMB(IMAGESIZE), DATABW(IMAGESIZE, tfrCPUtoGPU));
	printf("Kernel Execution    =%7.2f ms  ...  %4d MB  ...  %6.2f GB/s\n", kernelExecutionTime, DATAMB(GPUDataTransfer), DATABW(GPUDataTransfer, kernelExecutionTime));
	printf("GPU->CPU Transfer   =%7.2f ms  ...  %4d MB  ...  %6.2f GB/s\n", tfrGPUtoCPU, DATAMB(IMAGESIZE), DATABW(IMAGESIZE, tfrGPUtoCPU)); 
	printf("--------------------------------------------------------------------------\n");
	printf("Total time elapsed  =%7.2f ms       %4d MB  ...  %6.2f GB/s\n", totalTime, DATAMB((2*IMAGESIZE+GPUDataTransfer)), DATABW((2 * IMAGESIZE + GPUDataTransfer), totalTime));
	printf("--------------------------------------------------------------------------\n\n");

	// Deallocate CPU, GPU memory and destroy events.
	hipFree(imageG);
	hipFree(imgCpyG);
	hipEventDestroy(time1);
	hipEventDestroy(time2);
	hipEventDestroy(time3);
	hipEventDestroy(time4);
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Parallel Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		free(image);
		free(imgCpy);
		exit(EXIT_FAILURE);
	}
	free(image);
	free(imgCpy);
	return(EXIT_SUCCESS);
}




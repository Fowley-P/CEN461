#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <iostream>
#include <ctype.h>
#include <hip/hip_runtime.h>

#define MEGA 1024*1024
#define TWOBILLS 2000000000

typedef unsigned char uch;
typedef unsigned long ul;
typedef unsigned int  ui;

FILE *fw;
long num;
long count = 0;
int chunk, NumBlocks, NumThreads;
int *arri, *arriG;
int *arris;
float *arrf, *arrfG;
float *arrfs;
double *arrd, *arrdG;
double *arrds;
long *arrl, *arrlG;
long *arrls;

__global__
void *iBubG(int *arriG) {
	long myBid = blockIdx.x;
	long myTid = threadIdx.x;
	long ThPerBlk = blockDim.x;
	long myIndex = ((myBid*ThPerBlk)+myTid)*chunk;
	long i = myIndex, j = myIndex;
	int hold;
	for (; i<myIndex+chunk;i++) {
		for(; j<myIndex+chunk-(i-myIndex);j++) {
			if (arriG[j] < arriG[j+1]) {
				hold = arriG[j];
				arriG[j] = arriG[j+1];
				arriG[j+1] = hold;
			}
		}
	}
}
__global__
void *fBubG(float *arrfG) {
	long myBid = blockIdx.x;
	long myTid = threadIdx.x;
	long ThPerBlk = blockDim.x;
	long myIndex = ((myBid*ThPerBlk)+myTid)*chunk;
	long i = myIndex, j = myIndex;
	float hold;
	for (; i<myIndex+chunk;i++) {
		for(; j<myIndex+chunk-(i-myIndex);j++) {
			if (arrfG[j] < arrfG[j+1]) {
				hold = arrfG[j];
				arrfG[j] = arrfG[j+1];
				arrfG[j+1] = hold;
			}
		}
	}
}
__global__
void *dBubG(double *arrdG) {
	long myBid = blockIdx.x;
	long myTid = threadIdx.x;
	long ThPerBlk = blockDim.x;
	long myIndex = ((myBid*ThPerBlk)+myTid)*chunk;
	long i = myIndex, j = myIndex;
	double hold;
	for (; i<myIndex+chunk;i++) {
		for(; j<myIndex+chunk-(i-myIndex);j++) {
			if (arrdG[j] < arrdG[j+1]) {
				hold = arrdG[j];
				arrdG[j] = arrdG[j+1];
				arrdG[j+1] = hold;
			}
		}
	}
}
__global__
void *lBubG(long *arrlG) {
	long myBid = blockIdx.x;
	long myTid = threadIdx.x;
	long ThPerBlk = blockDim.x;
	long myIndex = ((myBid*ThPerBlk)+myTid)*chunk;
	long i = myIndex, j = myIndex;
	long hold;
	for (; i<myIndex+chunk;i++) {
		for(; j<myIndex+chunk-(i-myIndex);j++) {
			if (arrlG[j] < arrlG[j+1]) {
				hold = arri[j];
				arrlG[j] = arrlG[j+1];
				arrlG[j+1] = hold;
			}
		}
	}
}

void iMer() {
	int temp[chunk];
	int hold;
	long i;
	int j;
	while(count<num){
		i = count;
		j = 0;
		for (;i<count+chunk;i++)
			temp[i-count] = arri[i];
	
		for (i=0; i<count; i++) {
			while(temp[j] >= arri[i] && j<chunk-2) {
				j++;
			}
		
			if (temp[j] < arri[i]) {
				hold = temp[j];
				temp[j] = arri[i];
				arri[i] = hold;
			}
		}
		for (i = 0; i<chunk; i++)
			arri[i+count] = temp[i];

		count+=chunk;
	}
	
	return;
}

void fMer() {
	int temp[chunk];
	int hold;
	long i;
	int j;
	while(count<num){
		i = count;
		j = 0;
		for (;i<count+chunk;i++)
			temp[i-count] = arrf[i];
	
		for (i=0; i<count; i++) {
			while(temp[j] >= arrf[i] && j<chunk-2) {
				j++;
			}
		
			if (temp[j] < arrf[i]) {
				hold = temp[j];
				temp[j] = arrf[i];
				arrf[i] = hold;
			}
		}
		for (i = 0; i<chunk; i++)
			arrf[i+count] = temp[i];

		count+=chunk;
	}
	
	return;
}

void dMer() {
	int temp[chunk];
	int hold;
	long i;
	int j;
	while(count<num){
		i = count;
		j = 0;
		for (;i<count+chunk;i++)
			temp[i-count] = arri[i];
	
		for (i=0; i<count; i++) {
			while(temp[j] >= arrd[i] && j<chunk-2) {
				j++;
			}
		
			if (temp[j] < arrd[i]) {
				hold = temp[j];
				temp[j] = arrd[i];
				arrd[i] = hold;
			}
		}
		for (i = 0; i<chunk; i++)
			arrd[i+count] = temp[i];

		count+=chunk;
	}
	
	return;
}

void lMer() {
	int temp[chunk];
	int hold;
	long i;
	int j;
	while(count<num){
		i = count;
		j = 0;
		for (;i<count+chunk;i++)
			temp[i-count] = arrl[i];
	
		for (i=0; i<count; i++) {
			while(temp[j] >= arrl[i] && j<chunk-2) {
				j++;
			}
		
			if (temp[j] < arrl[i]) {
				hold = temp[j];
				temp[j] = arrl[i];
				arrl[i] = hold;
			}
		}
		for (i = 0; i<chunk; i++)
			arrl[i+count] = temp[i];

		count+=chunk;
	}
	
	return;
}

void genI () {
	float			totalTime, tfrCPUtoGPU, tfrGPUtoCPU, kernelExecutionTime; // GPU code run times
	hipError_t		cudaStatus, cudaStatus2;
	hipEvent_t		time1, time2, time3, time4;
	hipDeviceProp_t	GPUprop;
	arri = (int *)malloc(num*sizeof(int));
	arris = (int *)malloc(num*sizeof(int));
	long i;
	int pn;
	for(i = 0; i< num; i++) {
		pn = rand()%2;
		if (pn == 0)
			arri[i] = rand();
			
		else
			arri[i] = -1*rand();
	}
	
	int NumGPUs = 0;
	hipGetDeviceCount(&NumGPUs);
	if (NumGPUs == 0){
		printf("No CUDA Device is available\n");
		exit(EXIT_FAILURE);
	}
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		exit(EXIT_FAILURE);
	}
	
	hipGetDeviceProperties(&GPUprop, 0);
	
	hipEventCreate(&time1);
	hipEventCreate(&time2);
	hipEventCreate(&time3);
	hipEventCreate(&time4);
	
	hipEventRecord(time1, 0);		// Time stamp at the start of the GPU transfer
	// Allocate GPU buffer for the input and output images
	cudaStatus = hipMalloc((int *)&arriG, num*sizeof(int));
	if (cudaStatus!=hipSuccess) {
		fprintf(stderr, "hipMalloc failed! Can't allocate GPU memory\n");
		exit(EXIT_FAILURE);
	}
	
	cudaStatus = hipMemcpy(arriG, arri, num*(sizeof(int)), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy  CPU to GPU  failed!\n");
		exit(EXIT_FAILURE);
	}

	hipEventRecord(time2, 0);
	iBubG <<<NumBlocks, NumThreads>>> (arriG);
	
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching the kernel!\n", cudaStatus);
		exit(EXIT_FAILURE);
	}
	hipEventRecord(time3, 0);

	// Copy output (results) from GPU buffer to host (CPU) memory.
	cudaStatus = hipMemcpy(arri, arriG, num*sizeof(int)), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy GPU to CPU  failed!\n");
		exit(EXIT_FAILURE);
	}
	hipEventRecord(time4, 0);

	hipEventSynchronize(time1);
	hipEventSynchronize(time2);
	hipEventSynchronize(time3);
	hipEventSynchronize(time4);

	hipEventElapsedTime(&totalTime, time1, time4);
	hipEventElapsedTime(&tfrCPUtoGPU, time1, time2);
	hipEventElapsedTime(&kernelExecutionTime, time2, time3);
	hipEventElapsedTime(&tfrGPUtoCPU, time3, time4);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Program failed after hipDeviceSynchronize()!\n");
		free(arri);
		free(arris);
		exit(EXIT_FAILURE);
	}
	
	hipFree(arriG);
	hipEventDestroy(time1);
	hipEventDestroy(time2);
	hipEventDestroy(time3);
	hipEventDestroy(time4);
	
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		free(arri);
		free(arris);
		exit(EXIT_FAILURE);
	}
	count = 0;
	iMer();
	
	memcpy(arris, arri, num*sizeof(int));
	fw = fopen("outi.txt", "w");
	for (i = 0; i<num; i++) {
		fprintf(fw, "%d\n", arris[i]);
	}
	fclose(fw);
	free(arri);
	free(arris);
	return;
}

void genF () {
float			totalTime, tfrCPUtoGPU, tfrGPUtoCPU, kernelExecutionTime; // GPU code run times
	hipError_t		cudaStatus, cudaStatus2;
	hipEvent_t		time1, time2, time3, time4;
	hipDeviceProp_t	GPUprop;
	arrf = (float *)malloc(num*sizeof(float));
	arrfs = (float *)malloc(num*sizeof(float));
	long i;
	int pn;
	for(i = 0; i< num; i++) {
		pn = rand()%2;
		if (pn == 0)
			arrf[i] = rand();
			
		else
			arrf[i] = -1*rand();
	}
	
	int NumGPUs = 0;
	hipGetDeviceCount(&NumGPUs);
	if (NumGPUs == 0){
		printf("No CUDA Device is available\n");
		exit(EXIT_FAILURE);
	}
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		exit(EXIT_FAILURE);
	}
	
	hipGetDeviceProperties(&GPUprop, 0);
	
	hipEventCreate(&time1);
	hipEventCreate(&time2);
	hipEventCreate(&time3);
	hipEventCreate(&time4);
	
	hipEventRecord(time1, 0);		// Time stamp at the start of the GPU transfer
	// Allocate GPU buffer for the input and output images
	cudaStatus = hipMalloc((float0 *)&arrfG, num*sizeof(float));
	if (cudaStatus!=hipSuccess) {
		fprintf(stderr, "hipMalloc failed! Can't allocate GPU memory\n");
		exit(EXIT_FAILURE);
	}
	
	cudaStatus = hipMemcpy(arriG, arri, num*(sizeof(int)), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy  CPU to GPU  failed!\n");
		exit(EXIT_FAILURE);
	}

	hipEventRecord(time2, 0);
	iBubG <<<NumBlocks, NumThreads>>> (arriG);
	
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching the kernel!\n", cudaStatus);
		exit(EXIT_FAILURE);
	}
	hipEventRecord(time3, 0);

	// Copy output (results) from GPU buffer to host (CPU) memory.
	cudaStatus = hipMemcpy(arrf, arrfG, num*sizeof(float)), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy GPU to CPU  failed!\n");
		exit(EXIT_FAILURE);
	}
	hipEventRecord(time4, 0);

	hipEventSynchronize(time1);
	hipEventSynchronize(time2);
	hipEventSynchronize(time3);
	hipEventSynchronize(time4);

	hipEventElapsedTime(&totalTime, time1, time4);
	hipEventElapsedTime(&tfrCPUtoGPU, time1, time2);
	hipEventElapsedTime(&kernelExecutionTime, time2, time3);
	hipEventElapsedTime(&tfrGPUtoCPU, time3, time4);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Program failed after hipDeviceSynchronize()!\n");
		free(arrf);
		free(arrfs);
		exit(EXIT_FAILURE);
	}
	
	hipFree(arrfG);
	hipEventDestroy(time1);
	hipEventDestroy(time2);
	hipEventDestroy(time3);
	hipEventDestroy(time4);
	
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		free(arrf);
		free(arrfs);
		exit(EXIT_FAILURE);
	}
	count = 0;
	fMer();
	
	memcpy(arrfs, arrf, num*sizeof(float));
	fw = fopen("outf.txt", "w");
	for (i = 0; i<num; i++) {
		fprintf(fw, "%f\n", arrfs[i]);
	}
	fclose(fw);
	free(arrf);
	free(arrfs);
	return;
}

void genD () {
float			totalTime, tfrCPUtoGPU, tfrGPUtoCPU, kernelExecutionTime; // GPU code run times
	hipError_t		cudaStatus, cudaStatus2;
	hipEvent_t		time1, time2, time3, time4;
	hipDeviceProp_t	GPUprop;
	arrd = (double *)malloc(num*sizeof(double));
	arrds = (double *)malloc(num*sizeof(double));
	long i;
	int pn;
	for(i = 0; i< num; i++) {
		pn = rand()%2;
		if (pn == 0)
			arrd[i] = rand();
			
		else
			arrd[i] = -1*rand();
	}
	
	int NumGPUs = 0;
	hipGetDeviceCount(&NumGPUs);
	if (NumGPUs == 0){
		printf("No CUDA Device is available\n");
		exit(EXIT_FAILURE);
	}
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		exit(EXIT_FAILURE);
	}
	
	hipGetDeviceProperties(&GPUprop, 0);
	
	hipEventCreate(&time1);
	hipEventCreate(&time2);
	hipEventCreate(&time3);
	hipEventCreate(&time4);
	
	hipEventRecord(time1, 0);		// Time stamp at the start of the GPU transfer
	// Allocate GPU buffer for the input and output images
	cudaStatus = hipMalloc((double *)&arrdG, num*sizeof(double));
	if (cudaStatus!=hipSuccess) {
		fprintf(stderr, "hipMalloc failed! Can't allocate GPU memory\n");
		exit(EXIT_FAILURE);
	}
	
	cudaStatus = hipMemcpy(arriG, arri, num*(sizeof(int)), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy  CPU to GPU  failed!\n");
		exit(EXIT_FAILURE);
	}

	hipEventRecord(time2, 0);
	iBubG <<<NumBlocks, NumThreads>>> (arrdG);
	
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching the kernel!\n", cudaStatus);
		exit(EXIT_FAILURE);
	}
	hipEventRecord(time3, 0);

	// Copy output (results) from GPU buffer to host (CPU) memory.
	cudaStatus = hipMemcpy(arrd, arrdG, num*sizeof(double)), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy GPU to CPU  failed!\n");
		exit(EXIT_FAILURE);
	}
	hipEventRecord(time4, 0);

	hipEventSynchronize(time1);
	hipEventSynchronize(time2);
	hipEventSynchronize(time3);
	hipEventSynchronize(time4);

	hipEventElapsedTime(&totalTime, time1, time4);
	hipEventElapsedTime(&tfrCPUtoGPU, time1, time2);
	hipEventElapsedTime(&kernelExecutionTime, time2, time3);
	hipEventElapsedTime(&tfrGPUtoCPU, time3, time4);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Program failed after hipDeviceSynchronize()!\n");
		free(arrd);
		free(arrds);
		exit(EXIT_FAILURE);
	}
	
	hipFree(arrdG);
	hipEventDestroy(time1);
	hipEventDestroy(time2);
	hipEventDestroy(time3);
	hipEventDestroy(time4);
	
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		free(arrd);
		free(arrds);
		exit(EXIT_FAILURE);
	}
	count = 0;
	dMer();
	
	memcpy(arrds, arrd, num*sizeof(double));
	fw = fopen("outd.txt", "w");
	for (i = 0; i<num; i++) {
		fprintf(fw, "%lf\n", arrds[i]);
	}
	fclose(fw);
	free(arrd);
	free(arrds);
	return;
}

void genL () {
	float			totalTime, tfrCPUtoGPU, tfrGPUtoCPU, kernelExecutionTime; // GPU code run times
	hipError_t		cudaStatus, cudaStatus2;
	hipEvent_t		time1, time2, time3, time4;
	hipDeviceProp_t	GPUprop;
	arrl = (long *)malloc(num*sizeof(long));
	arrls = (long *)malloc(num*sizeof(long));
	long i;
	int pn;
	for(i = 0; i< num; i++) {
		pn = rand()%2;
		if (pn == 0) {
			pn = (rand() << 16);
			arrl[i] = pn%TWOBILLS;
		}	
		else {
			pn = (rand() << 16);
			arrl[i] = -1*(pn%TWOBILLS);
		}
	}
	int NumGPUs = 0;
	hipGetDeviceCount(&NumGPUs);
	if (NumGPUs == 0){
		printf("No CUDA Device is available\n");
		exit(EXIT_FAILURE);
	}
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		exit(EXIT_FAILURE);
	}
	
	hipGetDeviceProperties(&GPUprop, 0);
	
	hipEventCreate(&time1);
	hipEventCreate(&time2);
	hipEventCreate(&time3);
	hipEventCreate(&time4);
	
	hipEventRecord(time1, 0);		// Time stamp at the start of the GPU transfer
	// Allocate GPU buffer for the input and output images
	cudaStatus = hipMalloc((long *)&arrlG, num*sizeof(long));
	if (cudaStatus!=hipSuccess) {
		fprintf(stderr, "hipMalloc failed! Can't allocate GPU memory\n");
		exit(EXIT_FAILURE);
	}
	
	cudaStatus = hipMemcpy(arrlG, arrl, num*(sizeof(long)), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy  CPU to GPU  failed!\n");
		exit(EXIT_FAILURE);
	}

	hipEventRecord(time2, 0);
	lBubG <<<NumBlocks, NumThreads>>> (arrlG);
	
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching the kernel!\n", cudaStatus);
		exit(EXIT_FAILURE);
	}
	hipEventRecord(time3, 0);

	// Copy output (results) from GPU buffer to host (CPU) memory.
	cudaStatus = hipMemcpy(arrl, arrlG, num*sizeof(long)), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy GPU to CPU  failed!\n");
		exit(EXIT_FAILURE);
	}
	hipEventRecord(time4, 0);

	hipEventSynchronize(time1);
	hipEventSynchronize(time2);
	hipEventSynchronize(time3);
	hipEventSynchronize(time4);

	hipEventElapsedTime(&totalTime, time1, time4);
	hipEventElapsedTime(&tfrCPUtoGPU, time1, time2);
	hipEventElapsedTime(&kernelExecutionTime, time2, time3);
	hipEventElapsedTime(&tfrGPUtoCPU, time3, time4);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Program failed after hipDeviceSynchronize()!\n");
		free(arrl);
		free(arrls);
		exit(EXIT_FAILURE);
	}
	
	hipFree(arrlG);
	hipEventDestroy(time1);
	hipEventDestroy(time2);
	hipEventDestroy(time3);
	hipEventDestroy(time4);
	
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		free(arrl);
		free(arrls);
		exit(EXIT_FAILURE);
	}
	count = 0;
	lMer();
	
	memcpy(arrls, arrl, num*sizeof(long));
	fw = fopen("outl.txt", "w");
	for (i = 0; i<num; i++) {
		fprintf(fw, "%ld\n", arrls[i]);
	}
	fclose(fw);
	free(arrl);
	free(arrls);
	return;
}
		
int main (int argc, char** argv) {
	char choice;
		
	if (argc != 5) {
		printf("Incorrect call. Correct call:\n");
		printf("./filename num type chunk threads\n");
		return 1;
	}
	
	choice = toupper(*argv[2]);
	chunk = atoi(argv[3]);
	NumThreads = atoi(argv[4]);
	num = (atoi(argv[1])*MEGA);
	NumBlocks = num/(NumThreads*chunk);
	if (num%chunk!=0) {
		printf("Please choose a number that is evenly divisible by your #chunks.\n");
		return 1;
	}
	
	switch (choice) {
		case 'I': genI(); break;
		case 'F': genF(); break;
		case 'D': genD(); break;
		case 'L': genL(); break;
		default: printf("Please revise your function call. Accepted inputs are:\n");
				 printf("d, i, l, f, D, I, L, F.\n");
				 return 1;
	}
	
	return(EXIT_SUCCESS);
}

